#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
  extern __shared__ float s_data[];

  // global thread(data) row index 
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  i = i + 1; //because the edge of the data is not processed
  
  // global thread(data) column index
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

  int sharedDim = blockDim.x+2;
  int ty = threadIdx.y+1;
  int tx = threadIdx.x+1;

  j = j + 1; //because the edge of the data is not processed  

  // check the boundary
  if( i >= width - 1|| j >= width - 1|| i < 1 || j < 1) return;

//we will only copy the contents first. there are three cases
//1 left boundary, 2 right boundary and 3 middle

     //case 1 middle
	s_data[ty * sharedDim + tx]=g_dataA[i * floatpitch + j]; //itself
	s_data[(ty-1) * sharedDim +  tx]= g_dataA[(i-1) * floatpitch +  j];//N
	s_data[(ty+1) *  sharedDim+  tx]=g_dataA[(i+1) * floatpitch +  j];//S

    //case 2 left boundary
        s_data[(ty-1) *  sharedDim+ (tx-1)]=g_dataA[(i-1) * floatpitch + (j-1)];//NW
        s_data[ ty    *  sharedDim+ (tx-1)]=g_dataA[ i    * floatpitch + (j-1)];//W
	s_data[(ty+1) *  sharedDim+ (tx-1)]=g_dataA[(i+1) * floatpitch + (j-1)];//SW
 
    //case 3 right boundary
        s_data[(ty-1) * sharedDim + (tx+1)] =g_dataA[(i-1) * floatpitch + (j+1)];//NE
        s_data[ ty    * sharedDim+ (tx+1)] =g_dataA[ i    * floatpitch + (j+1)];//E
        s_data[(ty+1) * sharedDim + (tx+1)] =g_dataA[(i+1) * floatpitch + (j+1)];//SE
 
  __syncthreads();//wait for all threads to finish

  g_dataB[i * floatpitch + j] = (
                0.2f*s_data[ty * sharedDim + tx] +               //itself
                0.1f*s_data[(ty-1) * sharedDim +  tx   ] +       //N
                0.1f*s_data[(ty-1) * sharedDim + (tx+1)] +       //NE
                0.1f*s_data[ ty    *  sharedDim+ (tx+1)] +       //E
                0.1f*s_data[(ty+1) * sharedDim + (tx+1)] +       //SE
                0.1f*s_data[(ty+1) *  sharedDim+  tx   ] +       //S
                0.1f*s_data[(ty+1) *  sharedDim+ (tx-1)] +       //SW
                0.1f*s_data[ ty    *  sharedDim+ (tx-1)] +       //W
                0.1f*s_data[(ty-1) *  sharedDim+ (tx-1)]         //NW
                ) *0.95f;
}//end kernel1

